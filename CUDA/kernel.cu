#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void colorConvert(Pixel *pixels, unsigned int width, unsigned int height) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;
	//if (x < width && y < height) {
		unsigned char r = pixels[offset].r;
		unsigned char g = pixels[offset].g;
		unsigned char b = pixels[offset].b;
		pixels[offset].r = 0.21f*r;
		pixels[offset].g = 0.71f*g;
		pixels[offset].b = 0.07f*b;
	//}
}

hipError_t colorConvertWithCuda(Pixel *pixels, unsigned int width, unsigned int height, unsigned int nrChannels)
{
	Pixel *pixels_dev;
	hipError_t cudaStatus;
	size_t size = width * height;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&pixels_dev, size * sizeof(Pixel));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(pixels_dev, pixels, size * sizeof(Pixel), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 gridSize(width, height);
	colorConvert <<<gridSize, 1>>> (pixels_dev, width, height);
	cudaStatus = hipMemcpy(pixels, pixels_dev, size * sizeof(Pixel), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy f1ailed!");
		goto Error;
	}

Error:
	hipFree(pixels_dev);

	return cudaStatus;
}

